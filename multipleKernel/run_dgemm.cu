#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include "../includes/error_checking.h"
#include "../includes/performance_result.h"
#include "../includes/performance_utils.h"
#include "../includes/matrix_mul_kernel.h"
#include "../includes/dgemm_functions.h"
#include "kernel_registry.h"
#include "performance_result.h"

// Function to allocate memory and initialize matrices
void allocateAndInitializeMatrices(double **h_A, double **h_B, double **h_C, double **d_A, double **d_B, double **d_C, int m, int k, int n) {
    size_t size_A = m * k * sizeof(double);
    size_t size_B = k * n * sizeof(double);
    size_t size_C = m * n * sizeof(double);

    // Allocate host memory
    *h_A = (double*)malloc(size_A);
    *h_B = (double*)malloc(size_B);
    *h_C = (double*)malloc(size_C);

    // Initialize matrices
    for(int i = 0; i < m * k; i++) (*h_A)[i] = 1.0;
    for(int i = 0; i < k * n; i++) (*h_B)[i] = 2.0;
    for(int i = 0; i < m * n; i++) (*h_C)[i] = 0.0;

    // Allocate device memory
    CHECK_CUDA(hipMalloc(d_A, size_A));
    CHECK_CUDA(hipMalloc(d_B, size_B));
    CHECK_CUDA(hipMalloc(d_C, size_C));
}

// Function to transfer data to device
void transferDataToDevice(double *h_A, double *h_B, double *d_A, double *d_B, int m, int k, int n, PerformanceResult &result) {
    size_t size_A = m * k * sizeof(double);
    size_t size_B = k * n * sizeof(double);

    hipDeviceSynchronize();
    auto transfer_start = getCurrentTime();
    CHECK_CUDA(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    auto transfer_end = getCurrentTime();

    result.transfer_to_device_time = calculateDurationInSeconds(transfer_start, transfer_end);
    result.bandwidth_to_device = ((size_A + size_B) / (1024.0 * 1024.0 * 1024.0)) / result.transfer_to_device_time;

    printf("H2D Transfer: %.2f ms (%.2f GB/s)\n",
           result.transfer_to_device_time * 1000,
           result.bandwidth_to_device);
}

// Function to run a kernel and measure its performance
void runKernel(void (*kernel)(const double*, const double*, double*, int, int, int, double, double), 
               const double *d_A, const double *d_B, double *d_C, 
               int m, int n, int k, dim3 numBlocks, dim3 threadsPerBlock, 
               const char* kernelName, KernelResult &kernel_result, double alpha, double beta) {
    auto compute_start = getCurrentTime();
    kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k, alpha, beta);
    hipDeviceSynchronize();
    auto compute_end = getCurrentTime();

    kernel_result.computation_time = calculateDurationInSeconds(compute_start, compute_end);
    double flops = 2.0 * m * n * k;
    kernel_result.gflops = (kernel_result.computation_time > 0) ? 
        (flops / (kernel_result.computation_time * 1e9)) : 0.0;

    printf("%s Computation: %.2f ms (%.2f GFLOPS)\n",
           kernelName, kernel_result.computation_time * 1000, kernel_result.gflops);
}

// Function to transfer data from device to host
void transferDataFromDevice(double *h_C, double *d_C, int m, int n, 
    const char* kernelName, KernelResult &kernel_result) {
size_t size_C = m * n * sizeof(double);

hipDeviceSynchronize();
auto transfer_back_start = getCurrentTime();
CHECK_CUDA(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));
hipDeviceSynchronize();
auto transfer_back_end = getCurrentTime();

kernel_result.transfer_from_device_time = calculateDurationInSeconds(transfer_back_start, transfer_back_end);
kernel_result.bandwidth_from_device = (size_C / (1024.0 * 1024.0 * 1024.0)) / 
                  kernel_result.transfer_from_device_time;

printf("D2H Transfer %s: %.2f ms (%.2f GB/s)\n", 
kernelName, 
kernel_result.transfer_from_device_time * 1000, 
kernel_result.bandwidth_from_device);
}

// Updated verifyResults function
void verifyResults(const double *h_A, const double *h_B, double *h_C, int m, int n, int k, double alpha, double beta) {
    double epsilon = 1e-6;  // Tolerance for floating-point comparison
    bool correct = true;
    int errors = 0;
    const int max_errors_to_print = 10;

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            double expected = 0.0;
            for (int p = 0; p < k; ++p) {
                expected += h_A[i * k + p] * h_B[p * n + j];
            }
            expected = alpha * expected + beta * h_C[i * n + j];

            double actual = h_C[i * n + j];
            if (std::abs(expected - actual) > epsilon) {
                if (errors < max_errors_to_print) {
                    printf("Error at position (%d, %d): Expected %.8f, Got %.8f\n", i, j, expected, actual);
                }
                errors++;
                correct = false;
            }
        }
    }

    if (correct) {
        printf("Results verified: CORRECT\n");
    } else {
        printf("Results verified: INCORRECT. %d errors found.\n", errors);
    }
}

void runDGEMM(int size, std::vector<PerformanceResult>& results) {
    const int m = size;
    const int k = size;
    const int n = size;

    size_t total_memory = (m * k + k * n + m * n) * sizeof(double) / (1024.0 * 1024.0);
    printf("\nMatrix size: %d x %d (%zu MB)\n", size, size, total_memory);

    double *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    allocateAndInitializeMatrices(&h_A, &h_B, &h_C, &d_A, &d_B, &d_C, m, k, n);

    PerformanceResult result;
    result.size = size;

    transferDataToDevice(h_A, h_B, d_A, d_B, m, k, n, result);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    double alpha = 1.0;
    double beta = 0.0;

    // Get all registered kernels
    const auto& kernels = KernelRegistry::getInstance().getKernels();

    // Run each kernel
    for (const auto& kernel : kernels) {
        printf("----------------------------------------------- %s\n", kernel.name.c_str());
        
        // Reset C matrix
        for(int i = 0; i < m * n; i++) h_C[i] = 0.0;
        CHECK_CUDA(hipMemcpy(d_C, h_C, m * n * sizeof(double), hipMemcpyHostToDevice));

        KernelResult kernel_result;
        kernel_result.kernel_name = kernel.name;

        // Run kernel and measure performance
        runKernel(kernel.function, d_A, d_B, d_C, m, n, k, 
                 numBlocks, threadsPerBlock, kernel.name.c_str(), 
                 kernel_result, alpha, beta);

        // Transfer and measure results
        transferDataFromDevice(h_C, d_C, m, n, kernel.name.c_str(), kernel_result);

        // Verify results
        printf("Verifying %s results:\n", kernel.name.c_str());
        verifyResults(h_A, h_B, h_C, m, n, k, alpha, beta);

        // Store results
        result.kernel_results.push_back(kernel_result);
    }

    results.push_back(result);

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);
}